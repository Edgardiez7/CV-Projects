/*
 * PRACTICA DE CUDA 2020-2021
 * AUTOR:
 *
 *			EDGAR DIEZ ALONSO
 *
 * UNIVERSIDAD DE VALLADOLID, ETSI      
 */
/*
 * Simplified simulation of air flow in a wind tunnel
 *
 * CUDA version
 *
 * Computacion Paralela, Grado en Informatica (Universidad de Valladolid)
 * 2020/2021
 *
 * v1.4
 *
 * (c) 2021 Arturo Gonzalez Escribano
 */
 #include<stdio.h>
 #include<stdlib.h>
 #include<string.h>
 #include<math.h>
 #include<limits.h>
 #include<sys/time.h>
 
 /* Headers for the CUDA assignment versions */
 #include<hip/hip_runtime.h>
 
 #define	PRECISION	10000
 #define	STEPS		8
 
 /* Structure to represent a solid particle in the tunnel surface */
 typedef struct {
	 unsigned char extra;		// Extra field for student's usage
	 int pos_row, pos_col;		// Position in the grid
	 int mass;			// Particle mass
	 int resistance;			// Resistance to air flow
	 int speed_row, speed_col;	// Movement direction and speed
	 int old_flow;			// To annotate the flow before applying effects
 } Particle;
 
 
 /* 
  * Function to get wall time
  */
 double cp_Wtime(){
	 struct timeval tv;
	 gettimeofday(&tv, NULL);
	 return tv.tv_sec + 1.0e-6 * tv.tv_usec;
 }
 
 /* 
  * Macro function to simplify accessing with two coordinates to a flattened array
  * 	This macro-function can be changed and/or optimized by the students
  *
  */
 #define accessMat( arr, exp1, exp2 )	arr[ (int)(exp1) * columns + (int)(exp2) ]
 
 /*
  * Function: Update flow in a matrix position
  * 	This function can be changed and/or optimized by the students
  */
__host__ __device__ int update_flow( int *flow, int *flow_copy, int *particle_locations, int row, int col, int columns, int skip_particles ) {
	 // Skip update in particle positions
	 if ( skip_particles && accessMat( particle_locations, row, col ) != 0 ) return 0;
 
	 // Update in central part
	 if ( col > 0 && col < columns-1 ) {
		 accessMat( flow, row, col ) = 
			 ( 
			 accessMat( flow_copy, row, col ) + 
			 accessMat( flow_copy, row-1, col ) * 2 + 
			 accessMat( flow_copy, row-1, col-1 ) +
			 accessMat( flow_copy, row-1, col+1 ) 
			 ) / 5;
	 }
	 // Update if border left
	 else if ( col == 0 ) {
		 accessMat( flow, row, col ) = 
			 ( 
			 accessMat( flow_copy, row, col ) + 
			 accessMat( flow_copy, row-1, col ) * 2 + 
			 accessMat( flow_copy, row-1, col+1 ) 
			 ) / 4;
	 } 
	 // Update if border right
	 else {
		 accessMat( flow, row, col ) = 
			 ( 
			 accessMat( flow_copy, row, col ) + 
			 accessMat( flow_copy, row-1, col ) * 2 + 
			 accessMat( flow_copy, row-1, col-1 ) 
			 ) / 4;
	 }
	 
 
	 // Return flow variation at this position
	 return abs( accessMat( flow_copy, row, col ) - accessMat( flow, row, col ) );
 }
 
 
 /*
  * Function: Move particle
  * 	This function can be changed and/or optimized by the students
  */
 __host__ __device__ void move_particle( int *flow, Particle *particles, int particle, int rows, int columns ) {   // Compute movement for each step
	
    int step;
    for( step = 0; step < STEPS; step++ ) {
        // Highly simplified phisical model
        int row = particles[ particle ].pos_row / PRECISION;
        int col = particles[ particle ].pos_col / PRECISION;
        int pressure = accessMat( flow, row-1, col );
		/*if (particle == 6){
			printf("particula 6 con row=%d,col=%d,pression=%d.\n",row,col,pressure);			
		};*/
        int left, right;
        if ( col == 0 ) left = 0;
        else left = pressure - accessMat( flow, row-1, col-1 );
        if ( col == columns-1 ) right = 0;
        else right = pressure - accessMat( flow, row-1, col+1 );

        int flow_row = (int)( (float)pressure / particles[ particle ].mass * PRECISION );
        int flow_col = (int)( (float)(right - left) / particles[ particle ].mass * PRECISION );

        // Speed change
        particles[ particle ].speed_row =
            ( particles[ particle ].speed_row + flow_row ) / 2;
        particles[ particle ].speed_col =
            ( particles[ particle ].speed_col + flow_col ) / 2;

        // Movement
        particles[ particle ].pos_row =
            particles[ particle ].pos_row + particles[ particle ].speed_row / STEPS / 2;
        particles[ particle ].pos_col =
            particles[ particle ].pos_col + particles[ particle ].speed_col / STEPS / 2;

        // Control limits
        if ( particles[ particle ].pos_row >= PRECISION * rows )
            particles[ particle ].pos_row = PRECISION * rows - 1;
        if ( particles[ particle ].pos_col < 0 )
            particles[ particle ].pos_col = 0;
        if ( particles[ particle ].pos_col >= PRECISION * columns )
            particles[ particle ].pos_col = PRECISION * columns - 1;
    }
}
 
 
 #ifdef DEBUG
 /* 
  * Function: Print the current state of the simulation 
  */
 void print_status( int iteration, int rows, int columns, int *flow, int num_particles, int *particle_locations, int max_var ) {
	 /* 
	  * You don't need to optimize this function, it is only for pretty 
	  * printing and debugging purposes.
	  * It is not compiled in the production versions of the program.
	  * Thus, it is never used when measuring times in the leaderboard
	  */
	 int i,j;
	 printf("Iteration: %d, max_var: %f\n", 
		 iteration, 
		 (float)max_var / PRECISION
		 );
 
	 printf("  +");
	 for( j=0; j<columns; j++ ) printf("---");
	 printf("+\n");
	 for( i=0; i<rows; i++ ) {
		 if ( i % STEPS == iteration % STEPS )
			 printf("->|");
		 else
			 printf("  |");
 
		 for( j=0; j<columns; j++ ) {
			 char symbol;
			 if ( accessMat( flow, i, j )  >= 10 * PRECISION ) symbol = '*';
			 else if ( accessMat( flow, i, j ) >= 1 * PRECISION ) symbol = '0' + accessMat( flow, i, j ) / PRECISION;
			 else if ( accessMat( flow, i, j ) >= 0.5 * PRECISION ) symbol = '+';
			 else if ( accessMat( flow, i, j ) > 0 ) symbol = '.';
			 else symbol = ' ';
 
			 if ( accessMat( particle_locations, i, j ) > 0 ) printf("[%c]", symbol );
			 else printf(" %c ", symbol );
		 }
		 printf("|\n");
	 }
	 printf("  +");
	 for( j=0; j<columns; j++ ) printf("---");
	 printf("+\n\n");
 }
 #endif
 
 /*
  * Function: Print usage line in stderr
  */
 void show_usage( char *program_name ) {
	 fprintf(stderr,"Usage: %s ", program_name );
	 fprintf(stderr,"<rows> <columns> <maxIter> <threshold> <inlet_pos> <inlet_size> <fixed_particles_pos> <fixed_particles_size> <fixed_particles_density> <moving_particles_pos> <moving_particles_size> <moving_particles_density> <short_rnd1> <short_rnd2> <short_rnd3> [ <fixed_row> <fixed_col> <fixed_resistance> ... ]\n");
	 fprintf(stderr,"\n");
 }
 
 /*
  * KERNELS CUDA
 */
 __global__ void inlet(int* flowCuda, double* presionesCuda, double* noiseCuda, int columns, int inlet_size, int inlet_pos);
 __global__ void limpiar(int* particle_locationsCuda,int iter,int rows,int columns);
 __global__ void moveParticles(int* flowCuda, Particle* particlesCuda, int num_particles, int rows, int columns);
 __global__ void moveParticles2(int* particle_locationsCuda, Particle* particlesCuda, int num_particles, int columns);
 __global__ void updateFlow(int* flowCuda, int* flowcopyCuda, int* particle_locationsCuda, Particle* particlesCuda, int num_particles, int columns);
 __global__ void updateFlow2(int* flowCuda, int* particle_locationsCuda, Particle* particlesCuda, int num_particles, int columns);
 __global__ void almacenaMax(int* maxCuda,int* flowCuda,int* flow_copyCuda,int* particle_locationsCuda,int rows,int columns,int iter, int wave_front);
 __global__ void reductionMax(int* array, int size, int *result);
 __global__ void copiarFlow(int* flow_copyCuda, int* flowCuda, int iter, int rows, int columns);
 __global__ void resetMax(int* maxCuda, int tam);
 
 /*
  * MAIN PROGRAM
  */
 int main(int argc, char *argv[]) {
	 int i,j;
 
	 // Simulation data
	 int max_iter;			// Maximum number of simulation steps
	 int var_threshold;		// Threshold of variability to continue the simulation
	 int rows, columns;		// Cultivation area sizes
 
	 int *flow;			// Wind tunnel air-flow 
	 int *flow_copy;			// Wind tunnel air-flow (ancillary copy)
	 int *particle_locations;	// To quickly locate places with particles
 
	 int inlet_pos;			// First position of the inlet
	 int inlet_size;			// Inlet size
	 int particles_f_band_pos;	// First position of the band where fixed particles start
	 int particles_f_band_size;	// Size of the band where fixed particles start
	 int particles_m_band_pos;	// First position of the band where moving particles start
	 int particles_m_band_size;	// Size of the band where moving particles start
	 float particles_f_density;	// Density of starting fixed particles
	 float particles_m_density;	// Density of starting moving particles
 
	 unsigned short random_seq[3];		// Status of the random sequence
 
	 int		num_particles;		// Number of particles
	 Particle	*particles;		// List to store cells information
 
	 /* 1. Read simulation arguments */
	 /* 1.1. Check minimum number of arguments */
	 if (argc < 16) {
		 fprintf(stderr, "-- Error: Not enough arguments when reading configuration from the command line\n\n");
		 show_usage( argv[0] );
		 exit( EXIT_FAILURE );
	 }
 
	 /* 1.2. Read simulation area sizes, maximum number of iterations and threshold */
	 rows = atoi( argv[1] );
	 columns = atoi( argv[2] );
	 max_iter = atoi( argv[3] );
	 var_threshold = (int)(atof( argv[4] ) * PRECISION);
 
	 /* 1.3. Read inlet data and band of moving particles data */
	 inlet_pos = atoi( argv[5] );
	 inlet_size = atoi( argv[6] );
	 particles_f_band_pos = atoi( argv[7] );
	 particles_f_band_size = atoi( argv[8] );
	 particles_f_density = atof( argv[9] );
	 particles_m_band_pos = atoi( argv[10] );
	 particles_m_band_size = atoi( argv[11] );
	 particles_m_density = atof( argv[12] );
 
	 /* 1.4. Read random sequences initializer */
	 for( i=0; i<3; i++ ) {
		 random_seq[i] = (unsigned short)atoi( argv[13+i] );
	 }
 
	 /* 1.5. Allocate particles */
	 num_particles = 0;
	 // Check correct number of parameters for fixed particles
	 if (argc > 16 ) {
		 if ( (argc - 16) % 3 != 0 ) {
			 fprintf(stderr, "-- Error in number of fixed position particles\n\n");
			 show_usage( argv[0] );
			 exit( EXIT_FAILURE );
		 }
		 // Get number of fixed particles
		 num_particles = (argc - 16) / 3;
	 }
	 // Add number of fixed and moving particles in the bands
	 int num_particles_f_band = (int)( particles_f_band_size * columns * particles_f_density );
	 int num_particles_m_band = (int)( particles_m_band_size * columns * particles_m_density );
	 num_particles += num_particles_f_band;
	 num_particles += num_particles_m_band;
 
	 // Allocate space for particles
	 if ( num_particles > 0 ) {
		 particles = (Particle *)malloc( num_particles * sizeof( Particle ) );
		 if ( particles == NULL ) {
			 fprintf(stderr,"-- Error allocating particles structure for size: %d\n", num_particles );
			 exit( EXIT_FAILURE );
		 }
	 }
	 else particles = NULL;
 
	 /* 1.6.1. Read fixed particles */
	 int particle = 0;
	 if (argc > 16 ) {
		 int fixed_particles = (argc - 16) / 3;
		 for (particle = 0; particle < fixed_particles; particle++) {
			 particles[ particle ].pos_row = atoi( argv[ 16 + particle*3 ] ) * PRECISION;
			 particles[ particle ].pos_col = atoi( argv[ 17 + particle*3 ] ) * PRECISION;
			 particles[ particle ].mass = 0;
			 particles[ particle ].resistance = (int)( atof( argv[ 18 + particle*3 ] ) * PRECISION);
			 particles[ particle ].speed_row = 0;
			 particles[ particle ].speed_col = 0;
		 }
	 }
	 /* 1.6.2. Generate fixed particles in the band */
	 for ( ; particle < num_particles-num_particles_m_band; particle++) {
		 particles[ particle ].pos_row = (int)( PRECISION * ( particles_f_band_pos + particles_f_band_size * erand48( random_seq ) ) );
		 particles[ particle ].pos_col = (int)( PRECISION * columns * erand48( random_seq ) );
		 particles[ particle ].mass = 0;
		 particles[ particle ].resistance = (int)( PRECISION * erand48( random_seq ) );
		 particles[ particle ].speed_row = 0;
		 particles[ particle ].speed_col = 0;
	 }
 
	 /* 1.7. Generate moving particles in the band */
	 for ( ; particle < num_particles; particle++) {
		 particles[ particle ].pos_row = (int)( PRECISION * ( particles_m_band_pos + particles_m_band_size * erand48( random_seq ) ) );
		 particles[ particle ].pos_col = (int)( PRECISION * columns * erand48( random_seq ) );
		 particles[ particle ].mass = (int)( PRECISION * ( 1 + 5 * erand48( random_seq ) ) );
		 particles[ particle ].resistance = (int)( PRECISION * erand48( random_seq ) );
		 particles[ particle ].speed_row = 0;
		 particles[ particle ].speed_col = 0;
	 }
 
 #ifdef DEBUG
	 // 1.8. Print arguments 
	 printf("Arguments, Rows: %d, Columns: %d, max_iter: %d, threshold: %f\n", rows, columns, max_iter, (float)var_threshold/PRECISION );
	 printf("Arguments, Inlet: %d, %d  Band of fixed particles: %d, %d, %f  Band of moving particles: %d, %d, %f\n", inlet_pos, inlet_size, particles_f_band_pos, particles_f_band_size, particles_f_density, particles_m_band_pos, particles_m_band_size, particles_m_density );
	 printf("Arguments, Init Random Sequence: %hu,%hu,%hu\n", random_seq[0], random_seq[1], random_seq[2]);
	 printf("Particles: %d\n", num_particles );
	 for (int particle=0; particle<num_particles; particle++) {
		 printf("Particle[%d] = { %d, %d, %d, %d, %d, %d }\n",
				 particle,
				 particles[particle].pos_row, 
				 particles[particle].pos_col, 
				 particles[particle].mass, 
				 particles[particle].resistance, 
				 particles[particle].speed_row, 
				 particles[particle].speed_col
				 );
	 }
	 printf("\n");
 #endif // DEBUG
 
 
	 /* 2. Start global timer */
	 hipSetDevice(0);
	 hipDeviceSynchronize();
	 double ttotal = cp_Wtime();
 
 /*
  *
  * START HERE: DO NOT CHANGE THE CODE ABOVE THIS POINT
  *
  */	 
	 /* 3. Initialization */
	 int tam = sizeof(int) * (size_t)rows * (size_t)columns;
	 flow = (int *)malloc( tam );
	 flow_copy = (int *)malloc( tam );
	 particle_locations = (int *)malloc( tam );
	 int tamMax;
	 if(rows%STEPS==0)tamMax = rows * columns / STEPS;
	 else tamMax = ((rows/STEPS)+1) * columns;
	 int* max = (int *)malloc( sizeof(int) * tamMax);
	 int* maxVar=(int *)malloc( 1 * sizeof(int));
	 double* presiones = (double *)malloc(sizeof(double)*(size_t)inlet_size*4);
	 double* noise = (double *)malloc(sizeof(double)*(size_t)inlet_size);
 
	 if ( flow == NULL || flow_copy == NULL || particle_locations == NULL || max == NULL|| maxVar == NULL|| noise == NULL || presiones == NULL) {
		 fprintf(stderr,"-- Error allocating culture structures for size: %d x %d \n", rows, columns );
		 exit( EXIT_FAILURE );
	 }
 
	 for( i=0; i<rows; i++ ) {
		 for( j=0; j<columns; j++ ) {
			 accessMat( flow, i, j ) = 0;
			 accessMat( flow_copy, i, j ) = 0;
			 accessMat( particle_locations, i, j ) = 0;			 
		 }
	 }

	 for(i = 0; i < tamMax; i++){
		accessMat( max, 0, i) = 0;
	 }
 
	 int* flowCuda;
	 int* flow_copyCuda;
	 int* particle_locationsCuda;
	 int* maxCuda;
	 int* maxVarCuda;
	 double* presionesCuda;
	 double* noiseCuda;
	 Particle* particlesCuda;	
	 
	 
	 hipError_t error = hipSuccess;
	 hipMalloc(&flowCuda, tam);	
	 hipMalloc(&flow_copyCuda, tam);	
	 hipMalloc(&particle_locationsCuda, tam);	
	 hipMalloc(&maxCuda, sizeof(int)*tamMax);	
	 hipMalloc(&maxVarCuda, 1 * sizeof(int));
	 hipMalloc(&noiseCuda, (size_t)inlet_size * sizeof(double));
	 hipMalloc(&presionesCuda,(size_t)inlet_size * sizeof(double) * 4);
	 int tamParticles = num_particles * sizeof(Particle);
	 hipMalloc(&particlesCuda, tamParticles);
	 error=hipGetLastError();
	 if(error != hipSuccess){
		 printf("Cuda error allocating structures: %s\n", hipGetErrorString(error));
		 exit( EXIT_FAILURE );
	 }
	// printf("flow Posicion inicial: %p y final: %p.\n", flowCuda, &flowCuda[(tam/sizeof(int))-1]);
	 //printf("particles Posicion inicial: %p y final: %p.", particlesCuda, &particlesCuda[tamParticles-1] );
	 //kernel inlet	
	int gridx1;
	int threadnumber1=512;
	//if(inlet_size%threadnumber1==0 && inlet_size != 0) gridx1=inlet_size/threadnumber1;
	//else 
	gridx1=(inlet_size/threadnumber1)+1;	

	//kernel particulas
	int threadnumber3=512;
	int gridx3;			
	//if (num_particles%threadnumber3==0 && num_particles != 0) gridx3 = num_particles/threadnumber3;
	//else 
	gridx3=(num_particles/threadnumber3)+1;

	//kernel prop
	int gridyM, gridxM;	
	int threadnumber=16;
	//if(rows%threadnumber==0) gridyM=rows/(threadnumber*STEPS);
	//else 
	gridyM=(rows/(threadnumber*STEPS))+1;
	//if(columns%threadnumber==0) gridxM=columns/threadnumber;
	//else 
	gridxM=(columns/threadnumber)+1;	
	dim3 gridM(gridxM, gridyM);
	dim3 blockM(16,16);

	//kernel max
	int gridxmax;
	int threadnumberxmax=512;
	//if((rows*columns/STEPS)%threadnumberxmax==0) gridxmax=(rows*columns/STEPS)/threadnumberxmax;
	//else 
	gridxmax=((rows*columns/STEPS)/threadnumberxmax)+1;	
	
	hipMemcpy(flowCuda, flow, tam, hipMemcpyHostToDevice); 	
	hipMemcpy(particle_locationsCuda, particle_locations, tam, hipMemcpyHostToDevice); 
	hipMemcpy(particlesCuda, particles, tamParticles, hipMemcpyHostToDevice);	
	hipMemcpy(maxCuda, max, tam / STEPS, hipMemcpyHostToDevice);
	 
	 /* 4. Simulation */
	 int max_var = INT_MAX;
	 int iter;
	 for( iter=1; iter<=max_iter && max_var > var_threshold; iter++) {
		 //printf("entro en la iteracion. iter nº%d.\n", iter);
		
		 // 4.1. Change inlet values each STEP iterations
		 if ( iter % STEPS == 1 ) {
			double phase = iter / STEPS * ( M_PI / 4 );
			double phase_step = M_PI / 2 / inlet_size;
			 // 4.1.2. Add some random noise
			for(int i = 0; i<inlet_size; i++) noise[i]=  0.5 - erand48( random_seq );
			hipMemcpy(noiseCuda, noise, inlet_size * sizeof(double), hipMemcpyHostToDevice);
			// 4.1.1. Change the fans phase
			for(int i = 0; i<inlet_size*4; i++) presiones[i] = 9 + 2 * sin( phase + (i-inlet_pos) * phase_step );			
			hipMemcpy(presionesCuda, presiones, (size_t)inlet_size * sizeof(double)*4 , hipMemcpyHostToDevice);
			inlet<<<gridx1, threadnumber1>>>(flowCuda, presionesCuda, noiseCuda, columns, inlet_size, inlet_pos);
			/*error = cudaGetLastError();
			 if(error != cudaSuccess){
				 printf("Error inlet(): %s\n", cudaGetErrorString(error));
				 exit( EXIT_FAILURE );
			 }*/
			//cudaMemcpy(noise, noiseCuda, inlet_size * sizeof(double), cudaMemcpyDeviceToHost);
			//cudaMemcpy(presiones, presionesCuda, (size_t)inlet_size * sizeof(double)*4 , cudaMemcpyDeviceToHost);
			//cudaMemcpy(flow, flowCuda, tam, cudaMemcpyDeviceToHost);		
		 }  // End inlet update
		 	 
			
		 // 4.2. Particles movement each STEPS iterations
		 if ( iter % STEPS == 1 ) {
			 // Clean particle positions
			 int min;
			 (iter <= rows)? min = iter : min = rows;
			 int gridx2, gridy2;
			 int threadnumber2=16;
			 //if(min%threadnumber2==0 && min != 0) gridy2=min/threadnumber2;
			 //else 
			 gridy2=(min/threadnumber2)+1;
			 //if(columns%threadnumber2==0) gridx2=columns/threadnumber2;
			 //else 
			 gridx2=(columns/threadnumber2)+1;	
			 dim3 grid2(gridx2, gridy2);
			 dim3 block2(16,16);		
			limpiar<<<grid2,block2>>>(particle_locationsCuda, iter, rows, columns);
			/*error = cudaGetLastError();
			 if(error != cudaSuccess){
				 printf("Error limpiar(): %s\n", cudaGetErrorString(error));
				 exit( EXIT_FAILURE );
			 }*/
			//cudaMemcpy(particle_locations, particle_locationsCuda, tam, cudaMemcpyDeviceToHost);				 
					 
			//cudaMemcpy(flowCuda, flow, tam, cudaMemcpyHostToDevice);			
			moveParticles<<<gridx3, threadnumber3>>>(flowCuda, particlesCuda, num_particles, rows, columns);
			 /*error = cudaGetLastError();
			 if(error != cudaSuccess){
				 printf("Error moveParticles(): %s\n", cudaGetErrorString(error));
				 exit( EXIT_FAILURE );
			 }	*/		
			 //cudaMemcpy(flow, flowCuda, tam, cudaMemcpyDeviceToHost);
			 //cudaMemcpy(particles, particlesCuda, tamParticles, cudaMemcpyDeviceToHost);
 			 //cudaMemcpy(particlesCuda, particles, tamParticles, cudaMemcpyHostToDevice);	
			 //cudaMemcpy(particle_locationsCuda, particle_locations, tam, cudaMemcpyHostToDevice);	
			 moveParticles2<<<gridx3, threadnumber3>>>(particle_locationsCuda, particlesCuda, num_particles, columns);
			 /*error = cudaGetLastError();
			 if(error != cudaSuccess){
				 printf("Error moveParticles(): %s\n", cudaGetErrorString(error));
				 exit( EXIT_FAILURE );
			 }*/
			// cudaMemcpy( particles, particlesCuda, tamParticles, cudaMemcpyDeviceToHost);	
			//cudaMemcpy( particle_locations, particle_locationsCuda, tam, cudaMemcpyDeviceToHost);			 
		 
		 } // End particles movements
 
		
		 // 4.3. Effects due to particles each STEPS iterations
		 if ( iter % STEPS == 1 ) {

			//cudaMemcpy(flowCuda, flow, tam, cudaMemcpyHostToDevice);
			//cudaMemcpy(flow_copyCuda, flow_copy, tam, cudaMemcpyHostToDevice);
			//cudaMemcpy(particle_locationsCuda, particle_locations, tam, cudaMemcpyHostToDevice);
			//cudaMemcpy(particlesCuda, particles, tamParticles, cudaMemcpyHostToDevice);						
			updateFlow<<<gridx3, threadnumber3>>>(flowCuda, flow_copyCuda, particle_locationsCuda, particlesCuda, num_particles, columns);
			/*error = cudaGetLastError();
			if(error != cudaSuccess){
				printf("Error moveParticles(): %s\n", cudaGetErrorString(error));
				exit( EXIT_FAILURE );
			}	*/			
			//cudaMemcpy(flow, flowCuda, tam, cudaMemcpyDeviceToHost);
			//cudaMemcpy(flow_copy, flow_copyCuda, tam, cudaMemcpyDeviceToHost);
			//cudaMemcpy(particle_locations, particle_locationsCuda, tam, cudaMemcpyDeviceToHost);
			//cudaMemcpy(particles, particlesCuda, tamParticles, cudaMemcpyDeviceToHost);		



			//cudaMemcpy(flowCuda, flow, tam, cudaMemcpyHostToDevice);			
			//cudaMemcpy(particle_locationsCuda, particle_locations, tam, cudaMemcpyHostToDevice);
			//cudaMemcpy(particlesCuda, particles, tamParticles, cudaMemcpyHostToDevice);						
			updateFlow2<<<gridx3, threadnumber3>>>(flowCuda, particle_locationsCuda, particlesCuda, num_particles, columns);
			/*error = cudaGetLastError();
			if(error != cudaSuccess){
				printf("Error moveParticles(): %s\n", cudaGetErrorString(error));
				exit( EXIT_FAILURE );
			}	*/			
			//cudaMemcpy(flow, flowCuda, tam, cudaMemcpyDeviceToHost);			
			//cudaMemcpy(particle_locations, particle_locationsCuda, tam, cudaMemcpyDeviceToHost);
			//cudaMemcpy(particles, particlesCuda, tamParticles, cudaMemcpyDeviceToHost);				 
		 } // End effects
 		
		// 4.4. Copy data in the ancillary structure
		//cudaMemcpy(flowCuda, flow, tam, cudaMemcpyHostToDevice);
		//cudaMemcpy(flow_copyCuda, flow_copy, tam, cudaMemcpyHostToDevice);
		int min;
		(iter <= rows)? min = iter : min = rows;
		int gridy2, gridx2;
		int threadnumber2=16;
		//if(min%threadnumber2==0 && min != 0) gridy2=min/threadnumber2;
		//else 
		gridy2=(min/threadnumber2)+1;
		//if(columns%threadnumber2==0) gridx2=columns/threadnumber2;
		//else 
		gridx2=(columns/threadnumber2)+1;
		dim3 grid2(gridx2, gridy2);
		dim3 block2(16,16);		
		copiarFlow<<<grid2, block2>>>(flow_copyCuda, flowCuda, iter, rows, columns);
		error = hipGetLastError();
		/*if(error != cudaSuccess){
			printf("Error copiarFlow(): %s\n", cudaGetErrorString(error));
			exit( EXIT_FAILURE );
		}	*/	
		//cudaMemcpy(flow, flowCuda, tam, cudaMemcpyDeviceToHost);			
		//cudaMemcpy(flow_copy, flow_copyCuda, tam, cudaMemcpyDeviceToHost);
 		
		// 4.5. Propagation stage
		// 4.5.1. Initialize data to detect maximum variability
		
		if ( iter % STEPS == 1 ) { 			
			resetMax<<<gridxmax,threadnumberxmax>>>(maxCuda, tamMax);
			error = hipGetLastError();
			if(error != hipSuccess){
				printf("Error moveParticles(): %s\n", hipGetErrorString(error));
				exit( EXIT_FAILURE );
			}		
			maxVar[0]=0;
			max_var = 0;
			hipMemcpy(maxVarCuda, maxVar, sizeof(int), hipMemcpyHostToDevice);			
		}		
		// 4.5.2. Execute propagation on the wave fronts
		int wave_front = iter % STEPS;
		if ( wave_front == 0 ) wave_front = STEPS;			

	
		//cudaMemcpy(flowCuda, flow, tam, cudaMemcpyHostToDevice);
		//cudaMemcpy(maxCuda, max, tam, cudaMemcpyHostToDevice);		
		//cudaMemcpy(particle_locationsCuda, particle_locations, tam, cudaMemcpyHostToDevice);
			
		almacenaMax<<<gridM,blockM>>>(maxCuda, flowCuda, flow_copyCuda, particle_locationsCuda, rows, columns, iter,wave_front);
		error = hipGetLastError();
		/*if(error != cudaSuccess){
			printf("Error moveParticles(): %s\n", cudaGetErrorString(error));
			exit( EXIT_FAILURE );
		}*/
					
		//cudaMemcpy(particle_locations, particle_locationsCuda, tam, cudaMemcpyDeviceToHost);
		//cudaMemcpy(max, maxCuda, tam, cudaMemcpyDeviceToHost);			
		//cudaMemcpy(flow_copy, flow_copyCuda, tam, cudaMemcpyDeviceToHost);


		
		//cudaMemcpy(maxCuda, max, tam, cudaMemcpyHostToDevice);
		reductionMax<<<gridxmax,threadnumberxmax,sizeof(int)*threadnumberxmax>>>(maxCuda, rows*columns/STEPS, maxVarCuda);
		error = hipGetLastError();
		/*if(error != cudaSuccess){
			printf("Error moveParticles(): %s\n", cudaGetErrorString(error));
			exit( EXIT_FAILURE );
		}*/

		//cudaMemcpy(max, maxCuda, tam, cudaMemcpyDeviceToHost);
		hipMemcpy(maxVar, maxVarCuda, sizeof(int), hipMemcpyDeviceToHost);
		
		max_var = maxVar[0];	
		//printf("%d iteracion %d.\n", max_var, iter);		
		 
 #ifdef DEBUG
		 // 4.7. DEBUG: Print the current state of the simulation at the end of each iteration 
		 print_status( iter, rows, columns, flow, num_particles, particle_locations, max_var );
 #endif
 
	 } // End iterations
	
	 hipMemcpy(flow, flowCuda, tam, hipMemcpyDeviceToHost);

	 
	free(max);
	free(presiones);
	free(noise);
	hipFree(flowCuda);
	hipFree(flow_copyCuda);
	hipFree(particle_locationsCuda);
	hipFree(maxCuda);
	hipFree(maxVarCuda);
	hipFree(presionesCuda);
	hipFree(noiseCuda);
	hipFree(particlesCuda);
	

 /*
  *
  * STOP HERE: DO NOT CHANGE THE CODE BELOW THIS POINT
  *
  */
 
	 /* 5. Stop global timer */
	 hipDeviceSynchronize();
	 ttotal = cp_Wtime() - ttotal;
 
	 /* 6. Output for leaderboard */
	 printf("\n");
	 /* 6.1. Total computation time */
	 printf("Time: %lf\n", ttotal );
 
	 /* 6.2. Results: Statistics */
	 printf("Result: %d, %d", 
		 iter-1, 
		 max_var
		 );
	 int res_row = ( iter-1 < rows-1 ) ? iter-1 : rows-1;
	 int ind;
	 for ( ind=0; ind<6; ind++ )
		 printf(", %d", accessMat( flow, STEPS-1, ind * columns/6 ) );
	 for ( ind=0; ind<6; ind++ )
		 printf(", %d", accessMat( flow, res_row/2, ind * columns/6 ) );
	 for ( ind=0; ind<6; ind++ )
		 printf(", %d", accessMat( flow, res_row, ind * columns/6 ) );
	 printf("\n");
 
	 /* 7. Free resources */	
	 free( flow );
	 free( flow_copy );
	 free( particle_locations );
	 free( particles );
 
	 /* 8. End */
	 return 0;
 }

 __global__ void inlet(int* flowCuda, double* presionesCuda, double* noiseCuda, int columns, int inlet_size, int inlet_pos){
	int globalPos = threadIdx.x + blockIdx.x * blockDim.x;
	if(globalPos < inlet_size){
		accessMat(flowCuda, 0, globalPos + inlet_pos) = (int)(PRECISION * (presionesCuda[(globalPos + inlet_pos)%(4*inlet_size)] + noiseCuda[globalPos]));
	}
}

__global__ void limpiar(int* particle_locationsCuda,int iter,int rows,int columns){
	int j = threadIdx.x + blockIdx.x * blockDim.x;
	int i = threadIdx.y + blockIdx.y * blockDim.y;			
	if(i<=iter && i<rows && j < columns) accessMat( particle_locationsCuda, i, j ) = 0;
}
 
 __global__ void moveParticles(int* flowCuda, Particle* particlesCuda, int num_particles, int rows, int columns){	 
	 int particle =threadIdx.x + blockIdx.x * blockDim.x;		 
	 if(particle<num_particles){			
		 int mass = particlesCuda[ particle ].mass;		 		
		 // Fixed particles
		 if ( mass != 0 ){				
			move_particle( flowCuda, particlesCuda, particle, rows, columns);

	}} 
 }
 
 __global__ void moveParticles2(int* particle_locationsCuda, Particle* particlesCuda, int num_particles, int columns){
	 int i = blockIdx.x * blockDim.x + threadIdx.x;
	 	if(i<num_particles){			
			atomicAdd(&accessMat( particle_locationsCuda, particlesCuda[ i ].pos_row / PRECISION, particlesCuda[ i ].pos_col / PRECISION ), 1);
	 }
 }
 
 __global__ void updateFlow(int* flowCuda, int* flowcopyCuda, int* particle_locationsCuda, Particle* particlesCuda, int num_particles, int columns){
	int i = blockIdx.x * blockDim.x + threadIdx.x;	
	if(i<num_particles){				
		int row = particlesCuda[ i ].pos_row / PRECISION;
		int col = particlesCuda[ i ].pos_col / PRECISION;
		update_flow( flowCuda, flowcopyCuda, particle_locationsCuda, row, col, columns, 0 );
		particlesCuda[ i ].old_flow = accessMat( flowCuda, row, col );
 	}
}

__global__ void updateFlow2(int* flowCuda, int* particle_locationsCuda, Particle* particlesCuda, int num_particles, int columns){
	int i = blockIdx.x * blockDim.x + threadIdx.x;	
	if(i<num_particles){
		int row = particlesCuda[ i ].pos_row / PRECISION;
		int col = particlesCuda[ i ].pos_col / PRECISION;
		int resistance = particlesCuda[ i ].resistance;

		int back = (int)( (long)particlesCuda[ i ].old_flow * resistance / PRECISION ) / accessMat( particle_locationsCuda, row, col );
		atomicSub(&accessMat( flowCuda, row, col ), back); 
	
		atomicAdd(&accessMat( flowCuda, row-1, col ), back / 2);

		if ( col > 0 )
			atomicAdd(&accessMat( flowCuda, row-1, col-1 ) , back / 4);
		else
			atomicAdd(&accessMat( flowCuda, row-1, col ) , back / 4);
		if ( col < columns-1 )
			atomicAdd(&accessMat( flowCuda, row-1, col+1 ) , back / 4);
		else
			atomicAdd(&accessMat( flowCuda, row-1, col ) , back / 4);
	}	
}

__global__ void copiarFlow(int* flow_copyCuda, int* flowCuda, int iter, int rows, int columns){
	int j = threadIdx.x + blockIdx.x * blockDim.x;
	int i = threadIdx.y + blockIdx.y * blockDim.y;	
	if (i < iter && i < rows && j < columns) accessMat( flow_copyCuda, i, j ) = accessMat( flowCuda, i, j );	
}

__global__ void resetMax(int* maxCuda, int tam){
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	if(x < tam){
		maxCuda[x]=0;
	}
}

//propagacion
 __global__ void almacenaMax(int* maxCuda,int* flowCuda,int* flow_copyCuda,int* particle_locationsCuda,int rows,int columns,int iter, int wave_front){
	
	int j = (threadIdx.x + blockIdx.x * blockDim.x) ;
	int i = (threadIdx.y + blockIdx.y * blockDim.y) * STEPS + wave_front;
	if(i < rows && j<columns&& i >= wave_front){
		//printf("ENTROentro con iter=%d en [%d][%d].\n", iter, x, y);
		int var;
		if(i%STEPS==wave_front%STEPS && i<=iter && i!=0 ){
			//printf("entro");
			var = update_flow( flowCuda, flow_copyCuda, particle_locationsCuda, i, j, columns, 1 );
		//if ( var > maxCuda[x*columns+y] ){
			//printf("detecto nuevo maximo= %d.\n", var);	
			maxCuda[(i/STEPS)*columns+j] = var;
		}
		//}	
	}
}


/*
 * CUDA block reduction
 * Obtain the maximum value of an array of integers
 *
 * Inputs: 
 *	Device pointer to an array of integers of any size
 *	Size of the array
 *	Device pointer to an integer array of 1 element to store the result
 * 
 * Launching parameters (3 parameters):
 *	One-dimensional grid with enough blocks
 *	Any valid block size
 *	Dynamic shared memory size equal to: sizeof(int) * block size
 *
 * Output:
 *	The maximum value of the array is stored in *result
 *
 * (c) 2021, Arturo Gonzalez-Escribano
 * Simplification for an assignment in a Parallel Computing course,
 * Computing Engineering Degree, Universidad de Valladolid
 * Academic year 2020/2021
 */
 __global__ void reductionMax(int* array, int size, int *result){
	// Compute the global position of the thread in the grid
	int globalPos = threadIdx.x + blockIdx.x * blockDim.x;

	// Shared memory: One element per thread in the block
	// Call this kernel with the proper third launching parameter
	extern __shared__ int buffer[ ];

	// Load array values in the shared memory (0 if out of the array)
	if ( globalPos < size ) { 
		buffer[ threadIdx.x ] = array[ globalPos ];
	}
	else buffer[ threadIdx.x ] = 0;

	// Wait for all the threads of the block to finish
	__syncthreads();

	// Reduction tree
	for( int step=blockDim.x/2; step>=1; step /= 2 ) {
		if ( threadIdx.x < step )
			if ( buffer[ threadIdx.x ] < buffer[ threadIdx.x + step ] )
				buffer[ threadIdx.x ] = buffer[ threadIdx.x + step ];
		__syncthreads();
	}

	// The maximum value of this block is on the first position of buffer
	if ( threadIdx.x == 0 )
		atomicMax( result, buffer[0] );
}